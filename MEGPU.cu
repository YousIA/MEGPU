#include <iostream>
#include <hip/hip_runtime.h>

//#include <time.h>
//#include <cutil.h> 

using namespace std;
# define r 40
# define M 1000   // number of items
# define N  90 // number of transactions
# define alpha 1 // represents the weight of the support in the first fitness function
# define Beta 1 // represents the weight of the confidence in the first fitness function
# define k 15   // number of bees
struct ligne {int trans[N]; int nb;} *lg;
struct bee {int solution[N]; float cost;} *be;
/**************prototype declaration*******/
void read_trans(ligne T[]);// this function allows to read the transactional data base et insert it into the dataset vector
void display_dataset(ligne T[]); //this function allows to display the transactional data base
void display_solution(bee S); // this function display the current solution with its cost
float support_rule(ligne T[], int s[]); // this function calculates the support of the entire solution s
float support_antecedent(ligne T[], int s[]); // this function computes the support of the antecedent of the solution s
float confidence(int sr, int sa); // it calculates the confidence of the rule
float fitness1(int sr, int sa); // computes the fitness of a given solution s
void create_Sref(bee *s, ligne V[]); // here we create the solution reference sref and initialize it with the random way
bee neighborhood_computation(bee S, bee *V, ligne *D);// this function explores the local region for each bee
void search_area1(bee s, bee *T, int iteration, ligne V [],int flip); //detremines the search area for each bee using the first strategy
void search_area2(bee s, bee *T, int iteration, ligne V[], int flip); //detremines the search area for each bee using the second strategy
void search_area3(bee s, bee *T, int iteration, ligne V[], int distance); //detremines the search area for each bee using the third strategy
int W(int t[]); // indicates the  weight of solution representing by a vector t, this function is used on search_area3()
void copy(int t[], int v[]); // it copies the vector t in the vector v
int best_dance(bee *T); // return the best dance after the exploration of search region of each bee
void parallel_fitness(bee *V, ligne *D); // parallelize solution computing 
void display_bees(bee T[]); // display solutions
/*************************************************************************************/
__global__ void KernelSupport_rules(bee *N_List_GPU, int **compt_GPU, struct ligne *dataset_GPU){
	int thread_idx ;
	thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
	bool appartient=true;
    int indice=blockIdx.x*1000;
	indice=thread_idx-indice;

		int j=1;
		while (j<N){
			if (N_List_GPU[blockIdx.x].solution[indice]!=0){
				int l=0; 
				bool existe=false;
				while (l< dataset_GPU[thread_idx].nb && existe==false){
					if (dataset_GPU[thread_idx].trans[l]==j){
						existe=true;
					}
					l++;
				}
				if (existe==false){
					appartient=false;
				}
			}
			j++;    
		}
		if (appartient==true){
			//compt_GPU[blockIdx.x][thread_idx]=1;
		}
	
   // }
}

__global__ void KernelSupport_antecedent(bee *N_List_GPU, int **compt_GPU, struct ligne *dataset_GPU){
	int thread_idx ;
	thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
	bool appartient=true;
    int indice=blockIdx.x*M;
	indice=thread_idx-indice;
		int j=1;
		while (j<N){
			if (N_List_GPU[blockIdx.x].solution[indice]==1){
				int l=0; 
				bool existe=false;
				while (l< dataset_GPU[thread_idx].nb && existe==false){
					if (dataset_GPU[thread_idx].trans[l]==j){
						existe=true;
					}
					l++;
				}
				if (existe==false){
					appartient=false;
				}
			}
			j++;    
		}
		if (appartient==true){
			//compt_GPU[blockIdx.x][thread_idx]=1;
		}
	  
       //}
}
int main(void){
    FILE *f=NULL;
    f=fopen("/home/ydjenouri/mesprog/resultat1.txt","a");
    struct ligne *dataset_CPU, *dataset_GPU;
    struct bee *T_Dance;
	struct bee *N_List_CPU;
    struct bee Sref;
    struct bee best;
    int flip=1, distance, IMAX=1;
   // int k=5;
    hipEvent_t start, stop;
    float  elapsedTime;
    int j;
    /*****************************parallel program***********************/
	//allocation de la memoire dans le CPU
	dataset_CPU = (ligne *) malloc(M * sizeof(ligne)) ;
	T_Dance = (bee *) malloc(k * sizeof(bee)) ;
	N_List_CPU=(bee *) malloc(k *sizeof(bee));
       ////allocation de la memoire dans le GPU
       hipMalloc( (void**) &dataset_GPU, M*sizeof(ligne));
       
	   //read transactional database and insert in the dataset_CPU
       read_trans(dataset_CPU);
	   hipMemcpy(dataset_GPU, dataset_CPU, M * sizeof(ligne), hipMemcpyHostToDevice);
       hipEventCreate( &start );
       hipEventCreate( &stop );
       hipEventRecord( start, 0 ) ;
       create_Sref(&Sref, dataset_GPU); // creer une solution reference
	//display_solution(Sref);
      search_area1(Sref, T_Dance, IMAX, dataset_GPU, flip);
      printf("hello");
     // display_bees(T_Dance);
     // for ( int k=5; k<=15;k=k+5)
     for ( int i=0; i<=IMAX;i++)
	{
	    			
	    for ( j=0;j<k;j++) // neighborhood computation for all the solution in tab
					{ 
				       T_Dance[j]=neighborhood_computation(T_Dance[j], N_List_CPU, dataset_GPU);
					}
					/*j=best_dance(T_Dance,k);
					copy(T_Dance[j].solution,Sref.solution);
					Sref.cost=T_Dance[j].cost;
					if (Sref.cost > best.cost)//atte o maximisation
			    { 
					 copy(Sref.solution, best.solution);
					 best.cost=Sref.cost;
			    }
				*/
		       //display_bees(T_Dance);	 
//			//average=best.cost+average; 	        
			//printf("\nk="+b.k+" IMAX="+b.IMAX+"  average fitness="+average);
             search_area1(Sref,T_Dance, i, dataset_GPU, flip);
	
       } //Bso ending

    hipEventRecord( stop, 0 ) ;
    hipEventSynchronize( stop ) ;
    hipEventElapsedTime( &elapsedTime,start, stop ) ;
    printf("K=%d IMAX=%d  Execution Time in GPU : %3.1f ms\n", k,IMAX, elapsedTime );
   // fprintf(f,"K=%d IMAX=%d flip=%d Execution Time in GPU : %3.1f ms\n", k,flip,IMAX, elapsedTime );
    printf("Yes\n");
    hipEventDestroy( start );
    hipEventDestroy( stop );
//}// end loop IMAX
//} // end loop flip
//} // end loop k

//fclose(f);
hipFree(dataset_GPU);

return 0;
}
/**********************copry t in v********/
void copy(int t[], int v[])
{
for (int i=0;i<N; i++)
{
v[i]=t[i]; 
}     
}
/*******read transactional data bass and insert it in the data set structure********************************/
void read_trans(ligne T[]){
	char c='4';
	char t[100];
	int j;
	int i=0;
	int l=0;
	FILE *f=NULL;
	f=fopen("/home/ydjenouri/mesprog/T_90_1000.txt","r");
	if (f!=NULL) {
		//cout<<"the file is succefully opened"<<endl;
		j=0;
		while (c!=EOF){
			c=fgetc(f);
			if (c==' '){
				t[j]='\0';
				T[i].trans[l]=atoi(t);
                            l++;
				j=0;
			}
			if (c=='\n'){
				T[i].nb=l;
				l=0;
				i++;
				j=0;
			}
			if (c!=' ' && c!='\n'){
				t[j]=c;
				j++;
			}
		}   
		fclose(f);
	}
}
/*************************compute the support of the solution s**********/
float support_rule(ligne T[], int s[])
{
float compt=0;
		for (int i=0; i<M; i++)
		{
		bool appartient=true;
		
		int j=1;
		while (j<N)
		{
		 if (s[j]!=0)
		{
			int l=0; 
			bool existe=false;
			while (l< T[i].nb && existe==false)
			{
				if (T[i].trans[l]==j)
				{existe=true;}
			l++;
			}
			if (existe==false){appartient=false;}
		}
		j++;	
		}
		if (appartient==true) {compt++;}
		}
	   compt=compt/M;
	return compt;
}
/*****************************support antecedent computing*****************************/
float support_antecedent(ligne T[], int s[])
	{
             float compt=0;
		
		for (int i=0; i<M; i++)
		{
		bool appartient=true;
		int j=1;
		while (j<N)
		{
		 if (s[j]==1 ||s[j]==2)
		{
			int l=0; 
			bool existe=false;
			while (l< T[i].nb && existe==false)
			{
					if (T[i].trans[l]==j)
				        {existe=true;}
			l++;
			}
			if (existe==false){appartient=false;}
		}
		j++;	
		}
		if (appartient==true) {compt++;}
		}
	   compt=compt/M;
	//if(compt!=0)System.out.println("antecedent"+compt);
	   return compt;
	}
/****************************condifence computing**************************/
float confidence(int sr, int sa)
{
	float conf=1;
	conf=(float)sr/sa;
return conf;
}
/***********************evaluation of the solution s******/
float fitness1(int sr, int sa)
	{
	float cost=0; 
	//if (support_rule(sol)<Minsup || confidence(sol)<Minconf){cout=-1;}
	float x=(float)alpha*(sr/M);
	float y=(float)Beta*confidence(sr,sa);
	cost=x+y;
	return cost;
	}
/**************************display_solution*****************/
void display_solution(bee S)
{
for (int i=0;i<N;i++)
{
    printf("%d ", S.solution[i]);
}
printf ("cost is:%f",S.cost);
printf("\n");
}

/*********************create a solution reference Sref******************************************/
void create_Sref(bee *s, ligne V[])
{
	for (int i=0;i<N;i++){
		if (rand() % 2==0){
			(*s).solution[i]=0 ;
		}
		else {
			if (rand() % 2==0){
				(*s).solution[i]=0;
			}
			else {
				(*s).solution[i]=rand() % 3; 
			}
		}
	}
       //parallel_fitness(s, V);
}
/***********************************negihborhood computation************************/
bee neighborhood_computation(bee S, bee *V, ligne *D)
{
bee s;
int indice=0;
int i=0; 
bee neighbor, best_neighbor;
float best_cost=0;
		//copy(S.solution,best_neighbor);
		   copy(S.solution,neighbor.solution);
            while (i<k)
		   {
	          	    
	          if (neighbor.solution[indice]==0) 
	          {
	        		  if (rand()%2==0)
	        		  {neighbor.solution[indice]=1;}
	        		  else{neighbor.solution[indice]=2;}
	          }
	          else {
	          if (neighbor.solution[indice]==1) 
	          {
	        	  if (rand()%2==0)
                         neighbor.solution[indice]=0;

	        	  else {
	        	 neighbor.solution[indice]=2;
	        		  }
	          }
	          else {
	          if (neighbor.solution[indice]==2) 
	          {
	        	  	  if (rand()%2==0)
                                neighbor.solution[indice]=0;

	        	  else {
	                neighbor.solution[indice]=1;
	        		 }
	          }
	          }
	          }
	     indice++;
	     if (indice>=N){indice=0;}   
	     copy(neighbor.solution,V[i].solution);
         i++;
         
		 /*if (neighbor.cost>best_cost){copy(neighbor.solution,best_neighbor.solution);
                                            best_cost=neighbor.cost;}*/
		 }
		 parallel_fitness(V, D); 
		 
//copy(best_neighbor.solution, s.solution);
//s.cost=best_cost;
s.cost=0;
return s;
}
/************************determination of search area********************/
void search_area1(bee s, bee *T, int iteration, ligne V[],int flip)
{
	 
	    int indice=iteration % N;
	    int i=0;
		   while (i<k)
		   {
			   for (int j=0;j<N;j++)
			   {   
			    T[i].solution[j]=s.solution[j];	    
			   }	
	                 if (T[i].solution[indice]==0) 
	                {
	        	         if (iteration%4==0)
	        		  {T[i].solution[indice]=1;}
	        		  else{T[i].solution[indice]=2;}
	        		  
	        	    //           }    
	                }
			
	          else{
	          if (T[i].solution[indice]==1) 
	          {	  if (iteration%3==0)
			  {T[i].solution[indice]=0;}
			  else{T[i].solution[indice]=2;}
	        		 
	        }
	          else{ 
	          
	        	  if (iteration%2==0)
	    		  {
                       T[i].solution[indice]=1;}
	    		  else{
                         T[i].solution[indice]=0;}
	        	 }
	          }
	     indice=indice+flip;
	     if (indice>=N){indice=0;}   
		 parallel_fitness(&T[i], V);
		//T_Dance[i].cost=fitness1(T_Dance[i].solution);//evaluer solution  
		 i++;
		   }
}
/**************search 2*********************/
void search_area2(bee s, bee *T, int iteration, ligne V[], int flip)
{
int i=0;
int Nb_sol=0;
bool stop=false;
	  while (i<N && stop==false)
	  {
		   for (int j=0;j<N;j++)
		   {   
			   T[Nb_sol].solution[j]=s.solution[j];	       
		   }
		   for (int l=i;l<(i+flip)%N;l++)
		   {
		  if ( T[Nb_sol].solution[l]==0) 
	     {
	   	  if (rand()%2==1)
	   		  { T[Nb_sol].solution[l]=1;}
	   		  else{T[Nb_sol].solution[l]=2;}
	   		  
	   	}
	     else {
	     if (T[Nb_sol].solution[l]==1) 
	     {
	    	 if (rand()%2==1)
	  		  {T[Nb_sol].solution[l]=0;}
	  		  else{T[Nb_sol].solution[l]=2;}
	     }
	     else {
	       if (T[Nb_sol].solution[l]==2) 
	        {
	    	 if (rand()%2==0)
	  		  {T[Nb_sol].solution[l]=0;}
	  		  else{T[Nb_sol].solution[l]=1;}
	         }
	         }
	     }
		}
        parallel_fitness(&T[i], V);
	//T_Dance[Nb_sol].cost=fitness1(T_Dance[Nb_sol].solution); //evaluates the solution  
	Nb_sol++; 
	if (Nb_sol==k){stop=true;}   
	}
}   
/********search3***************************/
int W(int t[])
{
int w=0;
	for (int i=0;i<N; i++)
	{
	w=w+t[i];
	}
return w;
} 
/*******search 3 continued****************************/
void search_area3(bee s, bee *T, int iteration, ligne V[], int distance)
{
int Nb_sol=0;
	  while (Nb_sol!=k)
	  {
		   for (int j=0;j<N;j++)
		   {   
			  T[Nb_sol].solution[j]=s.solution[j];	 	    
		   }
		   int l=0;
		   int cpt=0;
		   while (cpt<distance)
		   {
		  if (T[Nb_sol].solution[l]==0) 
	     {
	   	  if (rand()%2==1)
	   		  {T[Nb_sol].solution[l]=1; cpt++;}
	   		  else{T[Nb_sol].solution[l]=2;cpt=cpt+2;}
	   		  
	   	}
	     else {
	     if (T[Nb_sol].solution[l]==1) 
	     {
	    	 if (rand()%2==0)
	  		  {T[Nb_sol].solution[l]=0;cpt++;}
	  		  else{T[Nb_sol].solution[l]=2;cpt++;}
	     }
	     else {
	       if (T[Nb_sol].solution[l]==2) 
	        {
	    	 if (rand()%2==0)
	  		  {T[Nb_sol].solution[l]=0;cpt=cpt+2;}
	  		  else{T[Nb_sol].solution[l]=1;cpt=cpt+1;}
	         }
	         }
	     
		   }
		  l=(l+1)%N;
		   } //end the small while
         //parallel_fitness(&T[Nb_sol], V);
	//T_Dance[Nb_sol].cost=fitness1(T_Dance[Nb_sol].solution);//assecees the solution  
	Nb_sol++; 
	  } // end the big while
 }
/********************************best dance********************/
int best_dance(bee *T)
{
	float max=T[0].cost;
	int indice=0;
	for (int i=1;i<k;i++)	
	{
     	if (T[i].cost>max)
	     {     
           max=T[i].cost;
		   indice=i;
         }
	}
return indice;
}
/***********************paralelize solution computing*******/
void parallel_fitness(bee *N_List_CPU, ligne V[])
{
bee *N_List_GPU;
	//int **compt;
    //compt = (int **) malloc(k*M*sizeof(int));
       	
		/*for (int i=0;i<k;i++){
		  for (int j=0;i<M;j++){
		compt[i][j]=0;
	}
	}*/
       int **compt_GPU;
	// cudaEventCreate( &start );
     	// cudaEventCreate( &stop );
     	// cudaEventRecord( start, 0 ) ;

	hipMalloc((void**) &N_List_GPU, k*sizeof(bee));
	//cudaMalloc( (void**) &compt_GPU, k*M* sizeof(int));
	hipMemcpy(N_List_GPU, N_List_CPU, k *sizeof(bee),hipMemcpyHostToDevice);
	//cudaMemcpy(compt_GPU, compt, k*M *sizeof(int),cudaMemcpyHostToDevice);

	KernelSupport_rules<<<20*N,M>>>(N_List_GPU, compt_GPU, V);
       //cudaMemcpy(compt, compt_GPU, k*M*sizeof(int),cudaMemcpyDeviceToHost);
       /*int sr=0; 
       for (int i=0;i<M;i++){
        	sr=sr+compt[i];
       }
       KernelSupport_antecedent<<<20*N,M>>>(s_GPU, compt_GPU, V);
       cudaMemcpy(compt, compt_GPU, M*sizeof(int),cudaMemcpyDeviceToHost);
       int sa=0; 
       for (int i=0;i<M;i++){
        	sa=sa+compt[i];
       }
       (*sol).cost=fitness1(sr,sa);*/
	   
}
/*****************************display T_dance************/
void display_bees(bee T[])
{
//FILE *f=NULL;
//f=fopen("/home/ydjenouri/mesprog/resultat1.txt","a");
//if (f!=NULL) {
for (int i=0;i<k;i++)
{
    for (int j=0;j<N;j++)
    {
    printf ("%d ",T[i].solution[j]);   
    }
    printf("%f", T[i].cost);
    printf("\n");
}
//fclose(f);
//}
}
